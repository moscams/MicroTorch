#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "cuda.hpp"
#include "ops.hpp"

namespace microtorch {

__global__ void fill_kernel(size_t n, float *self, float value) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < n; i += stride) {
    self[i] = value;
  }
}

template <>
void fill_impl<Cuda>(Tensor &self, const data_t value) {
  size_t blocks_per_grid = get_blocks_per_grid(self.numel());
  fill_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(self.numel(),
                                                    self.data_ptr(), value);
  CUDA_ERROR_CHECK();
}

__global__ void clone_kernel(size_t n, const float *a, float *out) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < n; i += stride) {
    out[i] = a[i];
  }
}

template <>
void clone_impl<Cuda>(const Tensor &a, Tensor &out) {
  float *a_ptr = a.data_ptr();
  float *out_ptr = out.data_ptr();

  size_t blocks_per_grid = get_blocks_per_grid(a.numel());
  clone_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(a.numel(), a_ptr, out_ptr);
  CUDA_ERROR_CHECK();
}

__global__ void clone_backward_kernel(size_t n, const float *grad_output_ptr,
                                      float *grad_input_ptr) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < n; i += stride) {
    grad_input_ptr[i] = grad_output_ptr[i];
  }
}

template <>
void clone_backward_impl<Cuda>(const Tensor &grad_output, Tensor &grad_input) {
  float *grad_output_ptr = grad_output.data_ptr();
  float *grad_input_ptr = grad_input.data_ptr();

  size_t blocks_per_grid = get_blocks_per_grid(grad_output.numel());
  clone_backward_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(
      grad_output.numel(), grad_output_ptr, grad_input_ptr);
  CUDA_ERROR_CHECK();
}

__global__ void rand_kernel(float *data, int numel) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  size_t stride = blockDim.x * gridDim.x;

  // For each block, initialize it's own state
  __shared__ hiprandState state;
  if (threadIdx.x == 0) {
    hiprand_init(1234, blockIdx.x, 0, &state);
  }
  __syncthreads();

  for (int i = idx; i < numel; i += stride) {
    data[i] = hiprand_uniform(&state);
  }
}

template <>
void rand_impl<Cuda>(Tensor &self) {
  size_t blocks_per_grid = get_blocks_per_grid(self.numel());
  rand_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(self.data_ptr(),
                                                    self.numel());
  CUDA_ERROR_CHECK();
}

}  // namespace microtorch
