#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "cuda.hpp"
#include "ops.hpp"

namespace microtorch {

__global__ void fill_kernel(int64_t n, float *self, float value) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t stride = blockDim.x * gridDim.x;
  for (int64_t i = idx; i < n; i += stride) {
    self[i] = value;
  }
}

template <>
void fill_impl<Cuda>(Tensor &self, const data_t value) {
  int64_t blocks_per_grid = get_blocks_per_grid(self.numel());
  fill_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(self.numel(),
                                                    self.data_ptr(), value);
  CUDA_ERROR_CHECK();
}

__global__ void clone_kernel(int64_t n, const float *a, float *out) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t stride = blockDim.x * gridDim.x;
  for (int64_t i = idx; i < n; i += stride) {
    out[i] = a[i];
  }
}

template <>
void clone_impl<Cuda>(const Tensor &a, Tensor &out) {
  float *a_ptr = a.data_ptr();
  float *out_ptr = out.data_ptr();

  int64_t blocks_per_grid = get_blocks_per_grid(a.numel());
  clone_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(a.numel(), a_ptr, out_ptr);
  CUDA_ERROR_CHECK();
}

__global__ void clone_backward_kernel(int64_t n, const float *grad_output_ptr,
                                      float *grad_input_ptr) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t stride = blockDim.x * gridDim.x;
  for (int64_t i = idx; i < n; i += stride) {
    grad_input_ptr[i] = grad_output_ptr[i];
  }
}

template <>
void clone_backward_impl<Cuda>(const Tensor &grad_output, Tensor &grad_input) {
  float *grad_output_ptr = grad_output.data_ptr();
  float *grad_input_ptr = grad_input.data_ptr();

  int64_t blocks_per_grid = get_blocks_per_grid(grad_output.numel());
  clone_backward_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(
      grad_output.numel(), grad_output_ptr, grad_input_ptr);
  CUDA_ERROR_CHECK();
}

__global__ void rand_kernel(float *data, int numel) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t stride = blockDim.x * gridDim.x;

  // For each block, initialize it's own state
  __shared__ hiprandState state;
  if (threadIdx.x == 0) {
    hiprand_init(1234, blockIdx.x, 0, &state);
  }
  __syncthreads();

  for (int i = idx; i < numel; i += stride) {
    data[i] = hiprand_uniform(&state);
  }
}

template <>
void rand_impl<Cuda>(Tensor &self) {
  int64_t blocks_per_grid = get_blocks_per_grid(self.numel());
  rand_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(self.data_ptr(),
                                                    self.numel());
  CUDA_ERROR_CHECK();
}

}  // namespace microtorch
