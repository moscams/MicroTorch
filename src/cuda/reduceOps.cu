#include "hip/hip_runtime.h"
#include "cuda.hpp"
#include "ops.hpp"

namespace microtorch {

__global__ void sum_kernel(int64_t n, float *a, float *out) {
  int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t stride = blockDim.x * gridDim.x;
  for (int64_t i = index; i < n; i += stride) {
    // Note: multi-thread write here, we should use atomicAdd
    atomicAdd(&out[0], a[i]);
  }
}

template <>
void sum_impl<Cuda>(Tensor &a, Tensor &out) {
  int64_t blocks_per_grid = get_blocks_per_grid(a.numel());
  sum_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(a.numel(), a.data_ptr(),
                                                   out.data_ptr());
  CUDA_ERROR_CHECK();
}

}  // namespace microtorch
