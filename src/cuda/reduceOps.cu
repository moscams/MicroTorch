#include "hip/hip_runtime.h"
#include "cuda.hpp"
#include "ops.hpp"

namespace microtorch {

__global__ void sum_kernel(int64_t n, float *a, float *out) {
  for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
       i += blockDim.x * gridDim.x) {
    // Note: multi-thread write here, we should use atomicAdd
    atomicAdd(&out[0], a[i]);
  }
}

template <>
void sum_impl<Cuda>(const Tensor &a, Tensor &out) {
  int64_t blocks_per_grid = get_blocks_per_grid(a.numel());
  sum_kernel<<<blocks_per_grid, ThreadsPerBlock>>>(a.numel(), a.data_ptr(),
                                                   out.data_ptr());
  CUDA_ERROR_CHECK();
}

}  // namespace microtorch
